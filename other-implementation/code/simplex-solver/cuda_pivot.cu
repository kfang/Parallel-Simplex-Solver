
#include <hip/hip_runtime.h>
//--------------------------------------------------------------------------
//  Cuda Pivot

__global__ void cuda_pivot(int pivot_row, int pivot_col,
		int num_rows, int num_cols,
		float* tableau)
{
	// Keep the pivot value in a register.
	float pivot_val = tableau[pivot_row*num_cols + pivot_col];

	int row = blockIdx.x;
	int col = threadIdx.x;

	// Calculate new value in tableau
	if (row != pivot_row) {
		float scale = tableau[row*num_cols + pivot_col]/pivot_val;
		tableau[row*num_cols + col] -= scale*tableau[pivot_row*num_cols + col];
	}

}

__global__ void cuda_test(int* vals) {
	*vals = 2;
}