#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------
//  Cuda Pivot

__global__ void cuda_pivot(int pivot_row, int pivot_col,
		int num_rows, int num_cols,
		float* tableau)
{
	// Keep the pivot value in a register.
	float pivot_val = tableau[pivot_row*num_cols + pivot_col];

	int row = blockIdx.x;
	int col = threadIdx.x;

	// Calculate new value in tableau
	if (row != pivot_row && col != pivot_col) {
		float scale = tableau[row*num_cols + pivot_col]/pivot_val;
		tableau[row*num_cols + col] -= scale*tableau[pivot_row*num_cols + col];
	}
	syncThreads();
	if (row != pivot_row && col == pivot_col) {
		tableau[row*num_cols + col] = 0.0;
	}

}

__global__ void cuda_test(int* vals) {
	*vals = 2;
}