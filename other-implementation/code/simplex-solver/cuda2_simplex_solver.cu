////////////////////////////////////////////////////////////////////////////
//                       serial_simplex_solver.cpp                        //
////////////////////////////////////////////////////////////////////////////


#include <iostream>
#include <cstdio>
#include <string>
#include "cuda2_simplex_solver.h"
#include "simplex_problem.h"
#include "simplex_solution.h"
#include "util.h"
#include "cuda_pivot.h"

//--------------------------------------------------------------------------
// CONSTRUCTORS AND DESTRUCTOR

Cuda2_Simplex_Solver::Cuda2_Simplex_Solver(void)
{
}

Cuda2_Simplex_Solver::~Cuda2_Simplex_Solver(void)
{
}

//--------------------------------------------------------------------------
// SOLVE

Simplex_Solution Cuda2_Simplex_Solver::solve(Simplex_Problem& problem)
{
	// Make a new tableau for solving the problem.
	float** tableau = create_tableau(problem);

	// Get the number of variables and constraints in the problem.
	int num_variables = problem.get_num_variables();
	int num_constraints = problem.get_num_constraints();

	// Calculate the number of rows and columns in the tableau and allocate memory.
	int num_rows = num_constraints + 1;
	int num_cols = num_variables + num_constraints + 1;

	double time = timestamp();

	//Cuda Pointer and mem
	float* cuda_row;
	float* cuda_pivot_row;

	//Make device space
	if (hipMalloc((void**)&cuda_row, num_cols*sizeof(float)) != hipSuccess) {
		std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cerr << "First Malloc Failed" << std::endl;
        exit(1);
	}
	if (hipMalloc((void**)&cuda_pivot_row, num_cols*sizeof(float)) != hipSuccess) {
		std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cerr << "Second Malloc Failed" << std::endl;
        exit(1);
	}



	// While the objective function can be increased, find a better
	// vertex on the simplex.
	int pivot_col, pivot_row;
	for (;;) {
		float min_val = tableau[0][0];
		pivot_col = 0;
		for (int i = 0; (i < num_cols-1); i++){
			if (tableau[0][i] < min_val) {
				min_val = tableau[0][i];
				pivot_col = i;
			}
		}
		for (pivot_row = 1; (pivot_row < num_rows) && (tableau[pivot_row][pivot_col] <= 0); pivot_row++);
		if (min_val >= 0) {
			break;
		}
		if (pivot_row >= num_rows) {
			//Then unbounded
			std::cout << "The problem is unbounded\n";
			return Simplex_Solution();
		}
		for (int i = pivot_row+1; i < num_rows; i++)
			if (tableau[i][pivot_col] > 0)
				if (tableau[i][num_cols-1]/tableau[i][pivot_col] < tableau[pivot_row][num_cols-1]/tableau[pivot_row][pivot_col])
					pivot_row = i;
		std::cerr << "---------------------------------" << std::endl;
		std::cerr << "BEFORE PIVOT" << std::endl;
		print_matrix(num_rows, num_cols, tableau);
		std::cerr << "pivot_row: " << pivot_row << std::endl;
		std::cerr << "pivot_col: " << pivot_col << std::endl;
		std::cerr << "AFTER PIVOT" << std::endl;
		pivot(pivot_row, pivot_col, num_rows, num_cols, tableau, cuda_row, cuda_pivot_row);
		print_matrix(num_rows, num_cols, tableau);
	}

	hipFree(cuda_row);
	hipFree(cuda_pivot_row);

	time = timestamp() - time;
	std::cerr << "Solve time: " << time << std::endl;

	std::cerr << "DONE!!!" << std::endl;
	std::cerr << "Max value: " << tableau[0][num_cols-1] << std::endl;

	std::cout << num_variables << "," << time << std::endl;

	return Simplex_Solution();
}

//--------------------------------------------------------------------------
// PIVOT

void Cuda2_Simplex_Solver::pivot(const int& pivot_row, const int& pivot_col,
                            const int& num_rows, const int& num_cols,
                            float** tableau, float* cuda_row, float* cuda_pivot_row)
{
	// Copy over pivot_row
	if (hipMemcpy(cuda_pivot_row, tableau[pivot_row], num_cols*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cerr << "Failed to copy row" << std::endl;
	exit(1);
	}

	float pivot_val = tableau[pivot_row][pivot_col];

	for(int i = 0; i < num_rows; i++) {
		if (i != pivot_row) {
			// Copy over row
			if (hipMemcpy(cuda_row, tableau[i], num_cols*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
				std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cerr << "Failed to copy row" << std::endl;
			exit(1);
			}

			float scale = tableau[i][pivot_col]/pivot_val;

			cuda2_pivot <<<(num_cols+127)/128, 128>>> (num_cols, scale, cuda_row, cuda_pivot_row);

			// Copy back
			if (hipMemcpy(tableau[i], cuda_row, num_cols*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
				std::cerr << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cerr << "Failed to copy back" << std::endl;
				exit(1);
			}
		}
	}

	// Scale the pivot row
	for (int col = 0; col < num_cols; col++) {
		tableau[pivot_row][col] /= pivot_val;
	}

}


//--------------------------------------------------------------------------
// CREATE_TABLEAU

float** Cuda2_Simplex_Solver::create_tableau(Simplex_Problem& problem)
{
	// Get the number of variables and constraints in the problem.
	int num_variables = problem.get_num_variables();
	int num_constraints = problem.get_num_constraints();

	// Calculate the number of rows and columns in the tableau and allocate memory.
	int num_rows = num_constraints + 1;
	int num_cols = num_variables + num_constraints + 1;
	float** tableau = create_2D_array<float>(num_rows, num_cols);

	// Add the objective function to the 0th row of the tableau.
	add_obj_func_to_tableau(num_rows, num_cols, tableau, problem);

	// Add the constraints to the rest of the rows of the tableau.
	add_constraints_to_tableau(num_rows, num_cols, tableau, problem);

	// The tableau is finished!
	return tableau;
}

//--------------------------------------------------------------------------
// ADD_OBJ_FUNC_TO_TABLEAU

void Cuda2_Simplex_Solver::add_obj_func_to_tableau(const int& num_rows,
                                                    const int& num_cols,
                                                    float** tableau,
                                                    Simplex_Problem& problem)
{
	int row = 0;
	int col = 0;
	for (Simplex_Problem::variable_iterator iter = problem.get_variable_iterator();
	     iter != problem.get_variable_end();
	     iter++)
	{
		const std::string& variable = *iter;
		float coeff = problem.get_obj_coeff(variable);
		if (coeff != 0) {
			tableau[row][col] = -1*coeff;
		}
		col++;
	}
}

//--------------------------------------------------------------------------
// ADD_CONSTRAINTS_TO_TABLEAU

void Cuda2_Simplex_Solver::add_constraints_to_tableau(const int& num_rows,
                                                       const int& num_cols,
                                                       float** tableau,
                                                       Simplex_Problem& problem)
{
	int row = 1;
	for (Simplex_Problem::constraint_name_iterator i = problem.get_constraint_name_iterator();
	     i != problem.get_constraint_name_end();
	     i++)
	{
		int col = 0;
		const std::string& constraint_name = *i;
		Constraint& constraint = problem.get_constraint(constraint_name);
		constraint_type type = constraint.get_type();
		for (Simplex_Problem::variable_iterator j = problem.get_variable_iterator();
		     j != problem.get_variable_end();
		     j++)
		{
			const std::string& variable = *j;
			float coeff = constraint.get_coefficient(variable);
			if (coeff != 0) {
				// This is a = or <= constraint.
				if (type == LEQ || type == EQ) {
					tableau[row][col] = coeff;
				}
				// This is a >= constraint, we multiply by -1 to make it <=.
				else {
					tableau[row][col] = -1*coeff;
				}
			}

			// Move to the next column/variable.
			col++;
		}
		// Add the slack variable term.
		tableau[row][col+row-1] = 1;

		// Add the right hand side of the equatioin.
		float rhs = constraint.get_rhs();
		if (type == LEQ || type == EQ) {
			tableau[row][num_cols-1] = rhs;
		} else {
			tableau[row][num_cols-1] = -1*rhs;
		}

		// Move to the next constraint.
		row++;
	}
}
